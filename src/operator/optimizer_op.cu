/*!
 *  Copyright (c) 2016 by Contributors
 * \file optimizer_op.cu
 * \brief Optimizer operators
 * \author Junyuan Xie
 */
#include "./optimizer_op-inl.h"

namespace mxnet {
namespace op {

NNVM_REGISTER_OP(sgd_update)
.set_attr<FCompute>("FCompute<gpu>", SGDUpdate<gpu>);

NNVM_REGISTER_OP(sgd_mom_update)
.set_attr<FCompute>("FCompute<gpu>", SGDMomUpdate<gpu>);

NNVM_REGISTER_OP(adam_update)
.set_attr<FCompute>("FCompute<gpu>", AdamUpdate<gpu>);

NNVM_REGISTER_OP(rmsprop_update)
.set_attr<FCompute>("FCompute<gpu>", RMSPropUpdate<gpu>);

}  // namespace op
}  // namespace mxnet
