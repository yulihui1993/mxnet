/*!
 *  Copyright (c) 2015 by Contributors
 * \file matrix_op.cu
 * \brief GPU Implementation of matrix operations
 */
// this will be invoked by gcc and compile GPU version
#include "./matrix_op-inl.h"
#include "./elemwise_unary_op.h"

namespace mxnet {
namespace op {
NNVM_REGISTER_OP(Reshape)
.set_attr<FCompute>("FCompute<gpu>", IdentityCompute<gpu>);

NNVM_REGISTER_OP(Flatten)
.set_attr<FCompute>("FCompute<gpu>", IdentityCompute<gpu>);

NNVM_REGISTER_OP(transpose)
.set_attr<FCompute>("FCompute<gpu>", Transpose<gpu>);

NNVM_REGISTER_OP(expand_dims)
.set_attr<FCompute>("FCompute<gpu>", IdentityCompute<gpu>);

NNVM_REGISTER_OP(crop)
.set_attr<FCompute>("FCompute<gpu>", Crop<gpu>);

NNVM_REGISTER_OP(_crop_assign)
.set_attr<FCompute>("FCompute<gpu>", CropAssign<gpu>);

NNVM_REGISTER_OP(_crop_assign_scalar)
.set_attr<FCompute>("FCompute<gpu>", CropAssignScalar<gpu>);

NNVM_REGISTER_OP(slice_axis)
.set_attr<FCompute>("FCompute<gpu>", Slice<gpu>);

NNVM_REGISTER_OP(_backward_slice_axis)
.set_attr<FCompute>("FCompute<gpu>", SliceGrad_<gpu>);

NNVM_REGISTER_OP(flip)
.set_attr<FCompute>("FCompute<gpu>", Flip<gpu>);

NNVM_REGISTER_OP(dot)
.set_attr<FCompute>("FCompute<gpu>", DotForward_<gpu>);

NNVM_REGISTER_OP(_backward_dot)
.set_attr<FCompute>("FCompute<gpu>", DotBackward_<gpu>);

NNVM_REGISTER_OP(batch_dot)
.set_attr<FCompute>("FCompute<gpu>", BatchDotForward_<gpu>);

NNVM_REGISTER_OP(_backward_batch_dot)
.set_attr<FCompute>("FCompute<gpu>", BatchDotBackward_<gpu>);

NNVM_REGISTER_OP(clip)
.set_attr<FCompute>("FCompute<gpu>", Clip<gpu>);

NNVM_REGISTER_OP(_backward_clip)
.set_attr<FCompute>("FCompute<gpu>", ClipGrad_<gpu>);

}  // namespace op
}  // namespace mxnet
